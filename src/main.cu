#include "hip/hip_runtime.h"
//
// Created by peter on 07/07/19.
//

#include <iostream>
#include <vector>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "utils.h"

__global__ void bitonic_sort_kernel(int *values, int j, int k) {
    unsigned  int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */

            if (values[i] > values[ixj]) {
                SWAP_INT(values[i], values[ixj]);
            }
        }

        if ((i & k) != 0) {
            /* Sort descending */

            if (values[i] < values[ixj]) {
                SWAP_INT(values[i], values[ixj]);
            }
        }
    }
}

void bitonic_sort(int *values, int size) {
    int threads = std::min(size, 512);
    int blocks = size / threads;

    std::cout << "Threads: " << threads << std::endl
              << "Blocks: " << blocks << std::endl
              << "Size: " << size << std::endl;


    int *dev_values;
    size_t dev_size = size * sizeof(int);

    hipMalloc((void **) &dev_values, dev_size);
    hipMemcpy(dev_values, values, dev_size, hipMemcpyHostToDevice);

    dim3 blocks_dim(blocks, 1);
    dim3 threads_dim(threads, 1);

    for (int i = 2; i <= size; i *= 2) {
        for (int j = i >> 1; j > 0; j /= 2) {
            bitonic_sort_kernel <<< blocks_dim, threads_dim, dev_size >>> (dev_values, j, i);
        }
    }

    hipMemcpy(values, dev_values, dev_size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

int main(int argc, char *argv[]) {
    // std::vector<int> elements = {3, 5, 8, 9, 7, 4, 2, 1};
    std::vector<int> elements = generate(1024, 0, 10000);

    // Check if size is power of two

    if (!is_power_of_two(elements.size())) {
        EXIT("Vector does not contain power of two n. of elements")
    }

    // CUDA sort

    int cuda_sorted[elements.size()];
    std::copy(elements.begin(), elements.end(), cuda_sorted);

    bitonic_sort(cuda_sorted, elements.size());

    // STD sort

    auto std_sorted = elements;

    std::sort(std_sorted.begin(), std_sorted.end());

    // Compare

    print(elements);
    print(cuda_sorted, elements.size());
    print(std_sorted);

    if (!compare(std_sorted, cuda_sorted)) {
        EXIT("CUDA and STD sorted does not match")
    } else {
        std::cout << "CUDA and STD sorts matched" << std::endl;
    }
}